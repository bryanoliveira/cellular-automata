#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernels.cuh"

////// DEVICE FUNCTIONS

__device__ inline unsigned short count_moore_neighbours(GridType *grid,
                                                        unsigned int rows,
                                                        unsigned int cols,
                                                        unsigned int idx) {
    unsigned short livingNeighbours = 0;
    // we can calculate the neighbours directly since we're using safety borders
    unsigned int neighbours[] = {
        idx - cols - 1, // top left
        idx - cols,     // top center
        idx - cols + 1, // top right
        idx - 1,        // middle left
        idx + 1,        // middle right
        idx + cols - 1, // bottom left
        idx + cols,     // bottom center
        idx + cols + 1, // bottom right
    };
    for (unsigned short nidx = 0; nidx < 8; nidx++)
        if (grid[neighbours[nidx]])
            livingNeighbours++;

    return livingNeighbours;
}

__device__ inline bool game_of_life(bool isAlive,
                                    unsigned short livingNeighbours) {
    // 1. Any live cell with two or three live neighbours survives.
    if (isAlive)
        return livingNeighbours == 2 || livingNeighbours == 3;
    // 2. Any dead cell with three live neighbours becomes a live cell.
    else if (livingNeighbours == 3)
        return true;
    // 3. All other live cells die in the next generation. Similarly,
    // all other dead cells stay dead.
    else
        return false;
}

////// CUDA KERNELS

__global__ void k_setup_rng(unsigned int rows, unsigned int cols,
                            hiprandState *globalRandState, unsigned long seed) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            hiprand_init(seed, idx, 0, &globalRandState[idx]);
        }
    }
}

__global__ void k_init_grid(GridType *grid, unsigned int rows,
                            unsigned int cols, hiprandState *globalRandState,
                            float spawnProbability) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            grid[idx] =
                hiprand_uniform(&globalRandState[idx]) < spawnProbability;
        }
    }
}

__global__ void k_update_grid_buffers(GridType *grid, vec2s *gridVertices,
                                      unsigned int rows, unsigned int cols) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            gridVertices[idx].state = (float)grid[idx];
        }
    }
}

__global__ void
k_update_grid_buffers_rescaled(GridType *grid, vec2s *gridVertices,
                               unsigned int rows, unsigned int cols,
                               unsigned int width, unsigned int height) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);
    float ppp = (rows * cols) / float(width * height);
    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            atomicAdd(&gridVertices[(y / height) * width + (x / width)].state,
                      float(grid[y * cols + x]) / ppp);
        }
    }
}

__global__ void k_reset_grid_buffers(vec2s *gridVertices, unsigned int width,
                                     unsigned int height) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < height - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < width - 1; x += stride.x) {
            unsigned int idx = y * width + x;
            gridVertices[idx].state = 0;
        }
    }
}

__global__ void k_compute_grid_count_rule(GridType *grid, GridType *nextGrid,
                                          unsigned int rows, unsigned int cols,
                                          hiprandState *globalRandState,
                                          float virtualSpawnProbability,
                                          bool countAliveCells,
                                          unsigned int *activeCellCount) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            bool newState = false;

            // 0. Virtual particle spawn probability
            if (virtualSpawnProbability > 0 &&
                hiprand_uniform(&globalRandState[idx]) < virtualSpawnProbability)
                newState = true;
            else
                newState = game_of_life(
                    grid[idx], count_moore_neighbours(grid, rows, cols, idx));

            // avoid atomicAdd when not necessary
            if (countAliveCells && newState)
                atomicAdd(activeCellCount, 1);

            nextGrid[idx] = newState;
        }
    }
}
