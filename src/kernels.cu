#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernels.cuh"

////// DEVICE FUNCTIONS

__device__ inline unsigned short count_moore_neighbours(Grid &grid,
                                                        unsigned int rows,
                                                        unsigned int cols,
                                                        unsigned int idx) {
    unsigned short livingNeighbours = 0;
    // we can calculate the neighbours directly since we're using safety borders
    unsigned int neighbours[] = {
        idx - cols - 1, // top left
        idx - cols,     // top center
        idx - cols + 1, // top right
        idx - 1,        // middle left
        idx + 1,        // middle right
        idx + cols - 1, // bottom left
        idx + cols,     // bottom center
        idx + cols + 1, // bottom right
    };
    for (unsigned short nidx = 0; nidx < 8; nidx++)
        if (grid[neighbours[nidx]])
            livingNeighbours++;

    return livingNeighbours;
}

__device__ inline bool game_of_life(bool isAlive,
                                    unsigned short livingNeighbours) {
    // 1. Any live cell with two or three live neighbours survives.
    if (isAlive)
        return livingNeighbours == 2 || livingNeighbours == 3;
    // 2. Any dead cell with three live neighbours becomes a live cell.
    else if (livingNeighbours == 3)
        return true;
    // 3. All other live cells die in the next generation. Similarly,
    // all other dead cells stay dead.
    else
        return false;
}

////// CUDA KERNELS

__global__ void k_setup_rng(unsigned int rows, unsigned int cols,
                            hiprandState *globalRandState, unsigned long seed) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            hiprand_init(seed, idx, 0, &globalRandState[idx]);
        }
    }
}

__global__ void k_init_grid(Grid &grid, unsigned int rows, unsigned int cols,
                            hiprandState *globalRandState,
                            float spawnProbability) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            grid[idx] =
                hiprand_uniform(&globalRandState[idx]) < spawnProbability;
        }
    }
}

__global__ void k_update_grid_buffers(Grid &grid, vec2s *gridVertices,
                                      unsigned int rows, unsigned int cols) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            gridVertices[idx].state = (float)grid[idx];
        }
    }
}

__global__ void k_compute_grid_count_rule(Grid &grid, Grid &nextGrid,
                                          unsigned int rows, unsigned int cols,
                                          hiprandState *globalRandState,
                                          float virtualSpawnProbability,
                                          bool countAliveCells,
                                          unsigned int *activeCellCount) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            bool newState = false;

            // 0. Virtual particle spawn probability
            if (virtualSpawnProbability > 0 &&
                hiprand_uniform(&globalRandState[idx]) < virtualSpawnProbability)
                newState = true;
            else
                newState = game_of_life(
                    grid[idx], count_moore_neighbours(grid, rows, cols, idx));

            // avoid atomicAdd when not necessary
            if (countAliveCells && newState)
                atomicAdd(activeCellCount, 1);

            nextGrid[idx] = newState;
        }
    }
}
