#include "automata_count_gpu.cuh"

namespace gpu {

CountAutomata::~CountAutomata() {}

void CountAutomata::run_evolution_kernel(bool countAliveCells) {
    k_compute_grid_count_rule<<<mGpuBlocks, mGpuThreadsPerBlock, 0,
                                mEvolveStream>>>(
        &grid[0], &nextGrid[0], config::rows, config::cols, mGlobalRandState,
        config::virtualFillProb, countAliveCells, mActiveCellCount);
    CUDA_ASSERT(hipGetLastError());
}

} // namespace gpu