#include "hip/hip_runtime.h"
// isolate cuda specific imports
#include <chrono>
#include <cuda_gl_interop.h>
#include <sstream>

#include "automata_base_gpu.cuh"

namespace gpu {

AutomataBase::AutomataBase(unsigned long randSeed,
                           std::ostringstream *const pLiveLogBuffer,
                           const unsigned int *gridVBO) {
    int gpuDeviceId;
    hipDeviceProp_t gpuProps;
    size_t gridSize = config::rows * config::cols;

    // define common kernel configs
    hipGetDevice(&gpuDeviceId);
    hipGetDeviceProperties(&gpuProps, gpuDeviceId);
    // blocks should be a multiple of #SMs on the grid (assume #SMs is even) !
    // actually the number of blocks should
    mGpuBlocks = dim3(gpuProps.multiProcessorCount / 2,
                      gpuProps.multiProcessorCount /
                          2); // 1156 on a 3080 - 17 blocks per SM
    // threads should be a multiple of warpSize on the block (assume warpSize is
    // even - it usually is 32)
    mGpuThreadsPerBlock = dim3(
        gpuProps.warpSize / 2,
        gpuProps.warpSize / 2); // 256 on a 3080 - 8 threads per SP (I think)

    // allocate memory
    CUDA_ASSERT(
        hipMalloc(&mGlobalRandState,
                   gridSize * sizeof(hiprandState))); // gpu only, not managed
    CUDA_ASSERT(hipMallocManaged(&mActiveCellCount, sizeof(unsigned int)));
    // instantiate grids
    grid = Grid(gridSize, true, false);
    nextGrid = Grid(gridSize, false);

    // initialize RNG
    k_setup_rng<<<mGpuBlocks, mGpuThreadsPerBlock>>>(
        config::rows, config::cols, mGlobalRandState, randSeed);

    // initialize grid
    k_init_grid<<<mGpuBlocks, mGpuThreadsPerBlock>>>(
        &grid, config::rows, config::cols, mGlobalRandState, config::fillProb);
    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());

    // create grid evolving CUDA stream
    CUDA_ASSERT(hipStreamCreate(&mEvolveStream));

    // if rendering is enabled
    if (gridVBO) {
        // create buffer updating CUDA stream
        CUDA_ASSERT(hipStreamCreate(&mBufferUpdateStream));
        // register OpenGL VBO to use with CUDA
        CUDA_ASSERT(hipGraphicsGLRegisterBuffer(
            &mGridVBOResource, *gridVBO, cudaGraphicsMapFlagsWriteDiscard));
    }

    // define the live log buffer
    mLiveLogBuffer = pLiveLogBuffer;
}

AutomataBase::~AutomataBase() {
    // wait for pending operations to complete
    CUDA_ASSERT(hipDeviceSynchronize());
    // destroy secondary streams
    hipStreamDestroy(mEvolveStream);
    hipStreamDestroy(mBufferUpdateStream);
    // unregister mapped resource (needs the GL context to still be set)
    // CUDA_ASSERT(hipGraphicsUnregisterResource(mGridVBOResource));
    // free the grid
    delete grid;
    delete nextGrid;
    CUDA_ASSERT(hipFree(mGlobalRandState));
}

void AutomataBase::compute_grid(bool logEnabled) {
    std::chrono::steady_clock::time_point timeStart;
    if (logEnabled)
        timeStart = std::chrono::steady_clock::now();

    *mActiveCellCount = 0; // this will be moved CPU<->GPU automatically
    run_evolution_kernel(logEnabled); // count alive cells if log is enabled
    // should I call hipDeviceSynchronize?
    CUDA_ASSERT(hipDeviceSynchronize());

    // simply swap buffers to avoid reallocation
    grid->swap(nextGrid);

    // calculate timings and update live buffer
    if (logEnabled)
        *mLiveLogBuffer << "| Evolve Kernel: "
                        << std::chrono::duration_cast<std::chrono::nanoseconds>(
                               std::chrono::steady_clock::now() - timeStart)
                               .count()
                        << " ns | Active cells: " << *mActiveCellCount << " |";
}

void AutomataBase::run_evolution_kernel(bool countAliveCells) {
    k_compute_grid_count_rule<<<mGpuBlocks, mGpuThreadsPerBlock, 0,
                                mEvolveStream>>>(
        &grid, &nextGrid, config::rows, config::cols, mGlobalRandState,
        config::virtualFillProb, countAliveCells, mActiveCellCount);
    CUDA_ASSERT(hipGetLastError());
}

void AutomataBase::update_grid_buffers() {
    if (!mGridVBOResource) {
        fprintf(stderr, "ERROR: Cannot call update_grid_buffers with rendering "
                        "disabled.\n");
        exit(EXIT_FAILURE);
    }

    // map OpenGL buffer object for writing from CUDA
    vec2s *gridVertices;
    CUDA_ASSERT(hipGraphicsMapResources(1, &mGridVBOResource, 0));
    size_t numBytes;
    CUDA_ASSERT(hipGraphicsResourceGetMappedPointer(
        (void **)&gridVertices, &numBytes, mGridVBOResource));
    // printf("CUDA mapped VBO: May access %ld bytes\n", numBytes);

    // launch kernel
    k_update_grid_buffers<<<mGpuBlocks, mGpuThreadsPerBlock, 0,
                            mBufferUpdateStream>>>(&grid, gridVertices,
                                                   config::rows, config::cols);

    CUDA_ASSERT(hipGetLastError());
    // should I call hipDeviceSynchronize?
    // CUDA_ASSERT(hipDeviceSynchronize());

    // unmap buffer object
    CUDA_ASSERT(hipGraphicsUnmapResources(1, &mGridVBOResource, 0));
}

} // namespace gpu